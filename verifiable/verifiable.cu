#include "hip/hip_runtime.h"
/*************************************************************************
 * Copyright (c) 2016-2022, NVIDIA CORPORATION. All rights reserved.
 * Modifications Copyright (c) 2020-2022 Advanced Micro Devices, Inc. All rights reserved.
 *
 * See LICENSE.txt for license information
 ************************************************************************/

//#pragma nv_diag_suppress declared_but_not_referenced

#include "verifiable.h"
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bfloat16.h>

#include "rccl/rccl.h"

#if NCCL_VERSION_CODE >= NCCL_VERSION(2,10,0) && RCCL_BFLOAT16 ==1
  #define HAVE_ncclBfloat16 1
#else
  #define HAVE_ncclBfloat16 0
#endif

#if NCCL_VERSION_CODE >= NCCL_VERSION(2,10,0)
  #define HAVE_ncclAvg 1
#else
  #define HAVE_ncclAvg 0
#endif

#if NCCL_VERSION_CODE >= NCCL_VERSION(2,11,0)
  #define HAVE_ncclPreMulSum 1
#else
  #define HAVE_ncclPreMulSum 0
#endif

#include <algorithm>
#include <cassert>
#include <cstdio>
#include <cstdint>
#include <cmath>
#include <unistd.h>

using std::size_t;
using std::int8_t;
using std::int16_t;
using std::int32_t;
using std::int64_t;
using std::uint8_t;
using std::uint16_t;
using std::uint32_t;
using std::uint64_t;

////////////////////////////////////////////////////////////////////////////////

namespace {
template<typename T>
__device__ unsigned long long bitsOf(T x) {
  union { unsigned long long ull; T val; } u;
  u.ull = 0;
  u.val = x;
  return u.ull;
}

__host__ __device__ uint64_t mixBits(uint64_t x) {
  union { uint32_t u32[2]; uint64_t u64; };
  u64 = x;
  u32[1] += 1;
  u32[0] ^= u32[1];
  u64 *= 0x9e3779b97f4a7c13u;
  u32[0] ^= u32[1]<<16 ^ u32[1]>>16;
  return u64;
}

__host__ __device__ uint64_t hashOf(uint64_t a, uint64_t b=0) {
  a += uint64_t(1)<<32;
  a += b;
  a ^= a>>32;
  a *= 0x9e3779b97f4a7c13u;
  a += b>>16 ^ b<<48;
  a ^= a>>32;
  a *= 0xc4ceb9fe1a85ec53u;
  return a;
}
}

////////////////////////////////////////////////////////////////////////////////

namespace {
template<typename T>
struct IsIntegral: std::is_integral<T> {};
template<>
struct IsIntegral<__half>: std::false_type {};
#if RCCL_BFLOAT16 == 1
template<>
struct IsIntegral<__hip_bfloat16>: std::false_type {};
#endif
}

////////////////////////////////////////////////////////////////////////////////

// Hide a value from arithmetic optimizations. Hopefully compiler cannot detect
// that this is equivalent to the identity function.
template<typename T>
__host__ __device__ T inhibit(T x) {
  union { uint64_t u64; T val; };
  u64 = 0;
  val = x;
  u64 *= 0x0000000100000001u;
  u64 *= 0xffffffff00000001u;
  return val;
}

////////////////////////////////////////////////////////////////////////////////

namespace {
  template<typename Y, typename X>
  __host__ __device__ Y castTo(X x) {
    return Y(x);
  }
  template<typename Y>
  __host__ __device__ Y castTo(float x) {
    return Y(x);
  }
  template<>
  __host__ __device__ __half castTo<__half>(float x) {
    return __float2half(x);
  }
  #if RCCL_BFLOAT16 == 1
  template<>
  __host__ __device__ __hip_bfloat16 castTo<__hip_bfloat16>(float x) {
    return hip_bfloat16(x);
  }
  #endif
}

////////////////////////////////////////////////////////////////////////////////
// The reduction functions

namespace {
struct ReduceNil {
  template<typename T>
  __host__ __device__ T preOp(T x, int /*rank_me*/) const { return x; }
  template<typename T>
  __host__ __device__ T operator()(T a, T /*b*/) const { return a; }
  template<typename T>
  __host__ __device__ T postOp(T x) const { return x; }
};
struct ReduceSum {
  template<typename T>
  __host__ __device__ T preOp(T x, int /*rank_me*/) const { return x; }
  template<typename T, typename=decltype(T()+T())>
  __host__ __device__ T operator()(T a, T b) const { return a + b; }
  __host__ __device__ __half operator()(__half a, __half b) const {
      return __float2half(__half2float(a) + __half2float(b));
  }
  #if RCCL_BFLOAT16 == 1
  __host__ __device__ __hip_bfloat16 operator()(__hip_bfloat16 a, __hip_bfloat16 b) const {
      return hip_bfloat16(static_cast<float>(a) + static_cast<float>(b));
  }
  #endif
  template<typename T>
  __host__ __device__ T postOp(T x) const { return x; }
};
struct ReduceProd {
  template<typename T>
  __host__ __device__ T preOp(T x, int /*rank_me*/) const { return x; }
  template<typename T, typename=decltype(T()*T())>
  __host__ __device__ T operator()(T a, T b) const { return a * b; }
  __host__ __device__ __half operator()(__half a, __half b) const {
      return __float2half(__half2float(a) * __half2float(b));
  }
  #if RCCL_BFLOAT16 == 1
  __host__ __device__ __hip_bfloat16 operator()(__hip_bfloat16 a, __hip_bfloat16 b) const {
      return hip_bfloat16(static_cast<float>(a) * static_cast<float>(b));
  }
  #endif
  template<typename T>
  __host__ __device__ T postOp(T x) const { return x; }
};
struct ReduceMin {
  template<typename T>
  __host__ __device__ T preOp(T x, int /*rank_me*/) const { return x; }
  template<typename T, typename=decltype(T()<T())>
  __host__ __device__ T operator()(T a, T b) const { return a < b ? a : b; }
  __host__ __device__ __half operator()(__half a, __half b) const {
    return __half2float(a) < __half2float(b) ? a : b;
  }
  #if RCCL_BFLOAT16 == 1
  __host__ __device__ __hip_bfloat16 operator()(__hip_bfloat16 a, __hip_bfloat16 b) const {
      return static_cast<float>(a) < static_cast<float>(b) ? a : b;
  }
  #endif
  template<typename T>
  __host__ __device__ T postOp(T x) const { return x; }
};
struct ReduceMax {
  template<typename T>
  __host__ __device__ T preOp(T x, int /*rank_me*/) const { return x; }
  template<typename T, typename=decltype(T()>T())>
  __host__ __device__ T operator()(T a, T b) const { return a > b ? a : b; }
  __host__ __device__ __half operator()(__half a, __half b) const {
      return __half2float(a) > __half2float(b) ? a : b;
  }
  #if RCCL_BFLOAT16 == 1
  __host__ __device__ __hip_bfloat16 operator()(__hip_bfloat16 a, __hip_bfloat16 b) const {
      return static_cast<float>(a) > static_cast<float>(b) ? a : b;
  }
  #endif
  template<typename T>
  __host__ __device__ T postOp(T x) const { return x; }
};
struct ReducePreMulSum {
  template<typename T>
  __host__ __device__ T preOp(T x, int rank_me) const {
    return ReduceProd()(x, ncclVerifiablePremulScalar<T>(rank_me));
  }
  template<typename T>
  __host__ __device__ T operator()(T a, T b) const { return ReduceSum()(a, b); }
  template<typename T>
  __host__ __device__ T postOp(T x) const { return x; }
};

template<typename T, bool integral = IsIntegral<T>::value>
struct ReduceAvg_Base;

template<typename T>
struct ReduceAvg_Base<T, /*integral=*/true> {
  int rank_n;
  __host__ __device__ T preOp(T x, int /*rank_me*/) const { return x; }
  __host__ __device__ T operator()(T a, T b) const { return ReduceSum()(a, b); }
  __host__ __device__ T postOp(T x) const { return x/rank_n; }
};

template<typename T>
struct ReduceAvg_Base<T, /*integral=*/false> {
  int rank_n;
  __host__ __device__ T preOp(T x, int /*rank_me*/) const {
    using T1 = typename std::conditional<(sizeof(T)<sizeof(double)), float, double>::type;
    return ReduceProd()(inhibit(castTo<T>(T1(1)/T1(rank_n))), inhibit(x));
  }
  __host__ __device__ T operator()(T a, T b) const { return ReduceSum()(a, b); }
  __host__ __device__ T postOp(T x) const { return x; }
};

struct ReduceAvg {
  int rank_n;
  template<typename T>
  __host__ __device__ T preOp(T x, int rank_me) const {
    return ReduceAvg_Base<T>{rank_n}.preOp(x, rank_me);
  }
  template<typename T>
  __host__ __device__ T operator()(T a, T b) const {
    return ReduceAvg_Base<T>{rank_n}(a, b);
  }
  template<typename T>
  __host__ __device__ T postOp(T x) const {
    return ReduceAvg_Base<T>{rank_n}.postOp(x);
  }
};
}

////////////////////////////////////////////////////////////////////////////////

namespace {
template<typename T>
struct FloatLayout;
template<>
struct FloatLayout<float> {
  static constexpr int exponent_bits = 8, mantissa_bits = 23;
  static constexpr int exponent_bias = (1<<(exponent_bits-1))-1;
};
template<>
struct FloatLayout<double> {
  static constexpr int exponent_bits = 11, mantissa_bits = 52;
  static constexpr int exponent_bias = (1<<(exponent_bits-1))-1;
};
template<>
struct FloatLayout<__half> {
  static constexpr int exponent_bits = 5, mantissa_bits = 10;
  static constexpr int exponent_bias = (1<<(exponent_bits-1))-1;
};
#if RCCL_BFLOAT16 == 1
template<>
struct FloatLayout<__hip_bfloat16> {
  static constexpr int exponent_bits = 8, mantissa_bits = 7;
  static constexpr int exponent_bias = (1<<(exponent_bits-1))-1;
};
#endif

template<typename T>
__host__ __device__ T makeFloat(int sign, int exp, uint64_t mant) {
  union { T ans; uint64_t bits; };
  bits = sign;
  bits <<= FloatLayout<T>::exponent_bits;
  bits |= exp;
  bits <<= FloatLayout<T>::mantissa_bits;
  bits |= mant;
  return ans;
}
}

////////////////////////////////////////////////////////////////////////////////

namespace {
// High bits of multiplcation are useful for generating bounded random values
// from unbounded random values. For instance, given X a totally random 32-bit
// integer, `umul32hi(X,n)` will be totally random within [0,n).
__host__ __device__ uint64_t umul32hi(uint32_t a, uint32_t b) {
#if HIP_VERSION > 50200000
  return __umulhi(a, b);
#else
  return uint64_t(a)*b >> 32;
#endif
}
__host__ __device__ uint64_t umul64hi(uint64_t a, uint64_t b) {
#if HIP_VERSION > 50200000
  return __umul64hi(a, b);
#else
  return uint64_t(__uint128_t(a)*__uint128_t(b) >> 64);
#endif
}

__host__ __device__ int clz32(int x) {
#if HIP_VERSION > 50200000
  return __clz(x);
#else
  return x==0 ? 32 : __builtin_clz(x);
#endif
}
__host__ __device__ int clz64(long long x) {
#if HIP_VERSION > 50200000
  return __clzll(x);
#else
  return x==0 ? 64 : __builtin_clzll(x);
#endif
}
}

////////////////////////////////////////////////////////////////////////////////

namespace {
// Returns a wildly permuted rank index. Useful when we know we want exactly N
// random ranks to exhibit some behavior, we can just test if:
// `shuffleRank(rank_n, rank_me, rng) < N`. Note that rank_n > 0 must be true
// for well defined results. This mixes the bits of rng.
__host__ __device__ int shuffleRank(int rank_n, int rank_me, uint64_t &rng) {
  uint32_t a = uint32_t(rng);
  uint32_t b = uint32_t(rng>>32);
  rng = mixBits(rng);

  uint32_t r = rank_me;
  // round down rank_n to largest pow2, then subtract 1
  uint32_t n2 = (~uint32_t(0)>>1) >> clz32(rank_n);

  // These are 1:1 functions modulo 2^n:
  //   f(x) = x*a + b : for odd a, any b
  //   f(x) = (x*x + x)/2
  // So we apply both to the bottom n2+1 ranks, then rotate the top
  // (rank_n-n2-1) to the bottom and apply both again.

  if(r <= n2) {
    // shuffle bottom n2+1 ranks
    r = (r*(a|1) + b) & n2;
    r = (r*r + r)/2 & n2;
    // rotate top to bottom
    r += rank_n - (n2+1);
  }
  else
    r -= n2+1; // rotate top to bottom

  if(r <= n2) {
    // shuffle bottom n2+1 again
    r = (r*(b|1) + a) & n2;
    r = (r*r + r)/2 & n2;
  }
  return r;
}
}

namespace {
// Generate wild integers x and y such that if every rank submits its x into a
// summation the result will be y with y <= y_max. Ranks should be shuffled
// before calling.
template<typename Uint>
__host__ __device__ void genSumXY(
    int rank_n, int rank_me, uint64_t &rng, Uint y_max, Uint &x, Uint &y,
    bool avoid_y=false // if true then returned y will not equal given y
  ) {
  static_assert(std::is_unsigned<Uint>::value, "Type must be unsigned integral.");

  { // Pick y as a random value in [y_max/2, y_max]
    Uint d, y_min = (y_max+1)/2;
    if(8*sizeof(Uint) > 32)
      d = umul64hi(rng, y_max/2 + (avoid_y ? 0 : 1));
    else
      d = umul32hi(uint32_t(rng), y_max/2 + (avoid_y ? 0 : 1));
    Uint y1 = (avoid_y ? y+1 : y_min) + d;
    y = y1 - (avoid_y && (y1 < y_min || y_max < y1) ? y_max/2 : 0);
  }
  rng = mixBits(rng);

  unsigned r = unsigned(rank_me);
  unsigned rn = unsigned(rank_n);
  // Partition our rn ranks into pn distinct subsets each of size rn/pn. If each
  // rank submits 1+p (where p is 0-based partition index) then the sum be:
  //   (rn/pn) * pn*(pn+1)/2
  // So set this equal to our desired sum y and solve for pn.
  //   (rn/pn) * pn*(pn+1)/2 = y
  //   rn*(pn+1)/2 = y
  //   pn = 2*(y/rn)-1
  Uint pn = rn == 1 ? 1 : 2*(y/rn) - 1;
  // In the case where rn is huge (compared to y) use only one partition meaning
  // that all rn ranks will submit 1 (since p=0).
  pn = pn == 0 ? 1 : pn;
  // Can't have more partitions than ranks.
  pn = rn < pn ? rn : pn;
  // Compute sum of contribution from pn partitions where each submits p+1.
  Uint p_sum;
  if(y_max <= ~uint32_t(0)>>1) // compile time known
    p_sum = Uint(uint32_t(pn)*uint32_t(pn+1)/2);
  else
    p_sum = Uint(uint64_t(pn)*uint64_t(pn+1)/2);
  // Let s be the number of ranks per partition. This is either rn/pn as we
  // intended, or y/p_sum if that's smaller to prevent overshooting our target y.
  uint32_t s = y/p_sum < rn/pn ? y/p_sum : rn/pn;
  x = (s != 0 && r/s < pn) ? 1 + r/s : 0; //  First s*pn ranks contribute partition index +1.
  x += r == rn-1 ? y - s*p_sum : 0; // Last rank contributes discrepancy.
}
}

namespace {
template<typename T>
__host__ __device__ T genInOutFloatSum(
    bool input_not_output, int rank_n, int rank_me, uint64_t seed, intptr_t index,
    bool same_sign
  ) {
  constexpr int exp_lo = 1 + FloatLayout<T>::mantissa_bits;
  constexpr int exp_hi = (1<<FloatLayout<T>::exponent_bits)-1;
  using uintmant_t = typename std::conditional<(8*sizeof(T) > 32), uint64_t, uint32_t>::type;
  constexpr uintmant_t mant_mask = (uintmant_t(1) << FloatLayout<T>::mantissa_bits)-1;
  constexpr uintmant_t max_mant = 2*mant_mask + 1; // add implicit leading 1
  uint64_t rng = hashOf(seed, index);

  int y_sign = rng & 1;
  int x_sign = y_sign;
  int xy_exp = exp_lo + umul32hi(uint32_t(rng>>32), exp_hi-exp_lo);
  rng = mixBits(rng);
  rank_me = shuffleRank(rank_n, rank_me, rng);

  // If we're using mixed signs then partition into evens and odds.
  int subrank_n = same_sign ? rank_n : (rank_n+1)/2;
  int subrank_me = same_sign ? rank_me : rank_me/2;
  uintmant_t x0_mant, y0_mant;
  genSumXY(subrank_n, subrank_me, rng, max_mant, x0_mant, y0_mant);

  if (!same_sign && (rank_n+0)/2 != 0) {
    uintmant_t x1_mant, y1_mant = y0_mant;
    // Avoid generating y1_mant == y0_mant so we don't have to worry about
    // signed zero as the result.
    genSumXY((rank_n+0)/2, rank_me/2, rng, max_mant, x1_mant, y1_mant, /*avoid_y=*/true);
    y_sign ^= y0_mant < y1_mant ? 1 : 0;
    y0_mant = (y0_mant < y1_mant ? -1 : 1)*(y0_mant - y1_mant);
    x_sign ^= rank_me%2;
    x0_mant = rank_me%2 == 0 ? x0_mant : x1_mant;
  }

  uintmant_t ans_mant = input_not_output ? x0_mant : y0_mant;
  if(ans_mant == 0)
    return T(0.0f);
  else {
    int shift = clz64(ans_mant) - (64-FloatLayout<T>::mantissa_bits-1);
    int ans_sign = input_not_output ? x_sign : y_sign;
    int ans_exp = xy_exp - shift;
    ans_mant <<= shift;
    return makeFloat<T>(ans_sign, ans_exp, ans_mant & mant_mask);
  }
}
}

namespace {
template<typename T>
__host__ __device__ T genInOutFloatPreMulSum(
    bool input_not_output, int rank_n, int rank_me, uint64_t seed, intptr_t index
  ) {
  constexpr int exp_lo = 1 + FloatLayout<T>::mantissa_bits;
  constexpr int exp_hi = (1<<FloatLayout<T>::exponent_bits)-1;
  using uintmant_t = typename std::conditional<(8*sizeof(T) > 32), uint64_t, uint32_t>::type;
  constexpr uintmant_t mant_mask = (uintmant_t(1) << FloatLayout<T>::mantissa_bits)-1;
  constexpr uintmant_t max_mant = 2*mant_mask + 1; // add implicit leading 1
  uint64_t rng = hashOf(seed, index);

  int y_sign = rng & 1;
  int y_exp = exp_lo + umul32hi(uint32_t(rng>>32), exp_hi-exp_lo);
  rng = mixBits(rng);
  int subrank_me0 = shuffleRank((rank_n+1)/2, rank_me/2, rng);
  int subrank_me1 = shuffleRank((rank_n+0)/2, rank_me/2, rng);

  // when ncclVerifiablePremulScalar() = 1.0 (rank_me%2 == 0)
  uintmant_t x0_mant, y0_mant;
  genSumXY((rank_n+1)/2, subrank_me0, rng, max_mant>>1, x0_mant, y0_mant);

  // when ncclVerifiablePremulScalar() = 2.0 (rank_me%2 == 1)
  uintmant_t x1_mant=0, y1_mant=0;
  if((rank_n+0)/2 != 0)
    genSumXY((rank_n+0)/2, subrank_me1, rng, max_mant>>2, x1_mant, y1_mant);

  uintmant_t x_mant = rank_me%2 == 0 ? x0_mant : x1_mant;
  uintmant_t y_mant = y0_mant + 2*y1_mant;
  uintmant_t ans_mant = input_not_output ? x_mant : y_mant;

  if(ans_mant == 0)
    return T(0.0f);
  else {
    int shift = clz64(ans_mant) - (64-FloatLayout<T>::mantissa_bits-1);
    int ans_sign = y_sign;
    int ans_exp = y_exp - shift;
    ans_mant <<= shift;
    return makeFloat<T>(ans_sign, ans_exp, ans_mant & mant_mask);
  }
}
}

namespace {
template<typename T>
__host__ __device__ T genInOutFloatProd(
    bool input_not_output, int rank_n, int rank_me, uint64_t seed, intptr_t index
  ) {
  // Three kinds of contributions (values for x):
  // 1) x = random value: only one rank does this
  // 2) x = 2^n: random positive n
  // 3) x = 1
  // Since only one rank submits a random value, the result of the product
  // will have the same mantissa as that value but with an exponent incorporating
  // the sum of the exponents from case (2)

  uint64_t rng = hashOf(seed, index);
  rank_me = shuffleRank(rank_n, rank_me, rng);
  int y_sign = (rank_n/2)%2;
  int x_sign = rank_me%2;

  constexpr unsigned max_exp = -1 + (1<<(FloatLayout<T>::exponent_bits-1));
  unsigned x_exp=0, y_exp=0;
  genSumXY(rank_n, rank_me, rng, max_exp, x_exp, y_exp);
  x_exp += FloatLayout<T>::exponent_bias;
  y_exp += FloatLayout<T>::exponent_bias;

  constexpr uint64_t mant_mask = (uint64_t(1)<<FloatLayout<T>::mantissa_bits)-1;
  uint64_t y_mant = rng & mant_mask;
  if (y_mant == 0) y_mant = 1;

  return makeFloat<T>(
    input_not_output ? x_sign : y_sign,
    input_not_output ? x_exp : y_exp,
    !input_not_output || rank_me==0 ? y_mant : 0
  );
}
}

////////////////////////////////////////////////////////////////////////////////
// What follows is lots of overloads for genInput/genOutput to generate data

namespace {
// General case for integral data for all ops but ReduceNil/premulsum
template<typename T, typename ReduceFn,
         typename = typename std::enable_if<
             !std::is_same<ReduceFn, ReduceNil>::value
           >::type>
__host__ __device__ void genInput(
    T &ans, ReduceFn, int rank_n, int rank_me, uint64_t seed, intptr_t index,
    std::true_type /*integral*/
  ) {
  (void)rank_n; // silence unused warnings
  union { uint64_t bits; T tmp; };
  bits = uint64_t(-1)>>(64 - 8*sizeof(T));
  bits &= hashOf(index ^ index<<16 ^ rank_me, seed);
  // make sure we never return 0 in products
  ans = std::is_same<ReduceFn, ReduceProd>::value && bits == 0 ? T(1) : tmp;
}
}

////////////////////////////////////////////////////////////////////////////////
// Dumb/generic case for genOutput just reduces results of genInput

namespace {
template<typename T, typename ReduceFn, bool IsIntegral>
__host__ __device__ void genOutput(
    T &ans, ReduceFn op, int rank_n, uint64_t seed, intptr_t index,
    std::integral_constant<bool, IsIntegral>
  ) {
  T acc = genInput<T>(op, rank_n, 0, seed, index);
  acc = op.preOp(acc, 0);
  for(int r=1; r < rank_n; r++)
    acc = op(acc, op.preOp(genInput<T>(op, rank_n, r, seed, index), r));
  ans = op.postOp(acc);
}
}

////////////////////////////////////////////////////////////////////////////////
// Nil reduction (byte copy functions). Optimized to assume rank_n=1

namespace {
template<typename T, bool IsIntegral>
__host__ __device__ void genInput(
    T &ans, ReduceNil, int rank_n, int rank_me, uint64_t seed, intptr_t index,
    std::integral_constant<bool, IsIntegral>
  ) {
  (void)rank_n, (void)rank_me; // silence unused warnings
  union { uint64_t bits; T tmp; };
  bits = mixBits(seed ^ index);
  bits >>= 64 - 8*sizeof(T);
  bits &= uint64_t(-1)>>(64 - 8*sizeof(T));
  ans = tmp;
}

template<typename T, typename ReduceFn, bool IsIntegral>
__host__ __device__ void genOutput(
    T &ans, ReduceNil op, int rank_n, uint64_t seed, intptr_t index,
    std::integral_constant<bool, IsIntegral>
  ) {
  ans = genInput<T>(op, rank_n, 0, seed, index);
}
}

////////////////////////////////////////////////////////////////////////////////
// Sum of float

namespace {
template<typename T>
__host__ __device__ void genInput(
    T &ans, ReduceSum, int rank_n, int rank_me, uint64_t seed, intptr_t index,
    std::false_type /*integral*/
  ) {
  ans = genInOutFloatSum<T>(/*input_not_output=*/true, rank_n, rank_me, seed, index, /*same_sign=*/false);
}

template<typename T>
__host__ __device__ void genOutput(
    T &ans, ReduceSum, int rank_n, uint64_t seed, intptr_t index,
    std::false_type /*integral*/
  ) {
  ans = genInOutFloatSum<T>(/*input_not_output=*/false, rank_n, 0, seed, index, /*same_sign=*/false);
}
}

////////////////////////////////////////////////////////////////////////////////
// Product of float

namespace {
template<typename T>
__host__ __device__ void genInput(
    T &ans, ReduceProd, int rank_n, int rank_me, uint64_t seed, intptr_t index,
    std::false_type /*integral*/
  ) {
  ans = genInOutFloatProd<T>(/*input_not_output=*/true, rank_n, rank_me, seed, index);
}

template<typename T>
__host__ __device__ void genOutput(
    T &ans, ReduceProd, int rank_n, uint64_t seed, intptr_t index,
    std::false_type /*integral*/
  ) {
  ans = genInOutFloatProd<T>(/*input_not_output=*/false, rank_n, 0, seed, index);
}
}

////////////////////////////////////////////////////////////////////////////////
// PreMulSum of int/float

namespace {
template<typename T>
__host__ __device__ void genInput(
    T &ans, ReducePreMulSum, int rank_n, int rank_me, uint64_t seed, intptr_t index,
    std::true_type integral
  ) {
  genInput(ans, ReduceSum(), rank_n, rank_me, seed, index, integral);
}

// No genOutput overload specific to premulsum(int), just use generic case.

template<typename T>
__host__ __device__ void genInput(
    T &ans, ReducePreMulSum, int rank_n, int rank_me, uint64_t seed, intptr_t index,
    std::false_type /*integral*/
  ) {
  ans = genInOutFloatPreMulSum<T>(/*input_not_output=*/true, rank_n, rank_me, seed, index);
}

template<typename T>
__host__ __device__ void genOutput(
    T &ans, ReducePreMulSum, int rank_n, uint64_t seed, intptr_t index,
    std::false_type /*integral*/
  ) {
  ans = genInOutFloatPreMulSum<T>(/*input_not_output=*/false, rank_n, 0, seed, index);
}
}

/////////////////////////////////////////////////////////////////////////////////
// Average of float

namespace {
template<typename T>
__host__ __device__ void genInput(
    T &ans, ReduceAvg, int rank_n, int rank_me, uint64_t seed, intptr_t index,
    std::false_type /*integral*/
  ) {
  ans = genInOutFloatSum<T>(/*input_not_output=*/true, rank_n, rank_me, seed, index, /*same_sign=*/true);
}

template<typename T>
__host__ __device__ void genOutput(
    T &ans, ReduceAvg, int rank_n, uint64_t seed, intptr_t index,
    std::false_type /*integral*/
  ) {
  ans = genInOutFloatSum<T>(/*input_not_output=*/false, rank_n, 0, seed, index, /*same_sign=*/true);
  using T1 = typename std::conditional<(sizeof(T)<sizeof(double)), float, double>::type;
  //ans = ReduceProd()(ans, T1(1)/T1(rank_n));
  ans = ReduceProd()(ans, inhibit(castTo<T>(T1(1)/T1(rank_n))));
 }
}

/////////////////////////////////////////////////////////////////////////////////
// min/max of float

namespace {
template<typename T>
__host__ __device__ void genInput(
    T &ans, ReduceMin, int rank_n, int rank_me, uint64_t seed, intptr_t index,
    std::false_type integral
  ) {
  genInput<T>(ans, ReduceMax(), rank_n, rank_me, seed, index, integral);
}
template<typename T>
__host__ __device__ void genInput(
    T &ans, ReduceMax, int rank_n, int rank_me, uint64_t seed, intptr_t index,
    std::false_type /*integral*/
  ) {
  (void)rank_n; // silence unused warnings
  constexpr uint64_t mant_mask = (uint64_t(1) << FloatLayout<T>::mantissa_bits)-1;
  uint64_t rng = hashOf(index ^ index<<16 ^ rank_me, seed);
  int sign = rng & 1;
  rng ^= rng>>1;
  int exp = rng & ((1<<(FloatLayout<T>::exponent_bits-1))-1);
  exp += 1<<(FloatLayout<T>::exponent_bits-2);
  rng ^= rng >> FloatLayout<T>::exponent_bits;
  uint64_t mant = rng & mant_mask;
  ans = makeFloat<T>(sign, exp, mant);
}

// No genOutput overload specific to floating point min/max, just use generic case.
}

///////////////////////////////////////////////////////////////////////////////
// Entry API for genInput/genOutput

namespace {
template<typename T, typename ReduceFn>
__host__ __device__ T genInput(
    ReduceFn op, int rank_n, int rank_me, uint64_t seed, intptr_t index
  ) {
  T ans;
  genInput(ans, op, rank_n, rank_me, seed, index,
    std::integral_constant<bool, IsIntegral<T>::value>());
  return ans;
}

template<typename T, typename ReduceFn>
__host__ __device__ T genOutput(
    ReduceFn op, int rank_n, uint64_t seed, intptr_t index
  ) {
  T ans;
  genOutput(ans, op, rank_n, seed, index,
    std::integral_constant<bool, IsIntegral<T>::value>());
  return ans;
}
}

////////////////////////////////////////////////////////////////////////////////

#if !SELF_TEST
namespace {
template<typename T, typename ReduceFn>
__global__ void prepareInput2(
    T *elts, intptr_t elt_n, ReduceFn op, int rank_n, int rank_me,
    uint64_t seed, intptr_t elt_ix0
  ) {
  intptr_t i0 = blockIdx.x*(elt_n/gridDim.x);
  i0 += blockIdx.x < elt_n%gridDim.x ? blockIdx.x : elt_n%gridDim.x;
  intptr_t i1 = (blockIdx.x+1)*(elt_n/gridDim.x);
  i1 += blockIdx.x+1 < elt_n%gridDim.x ? blockIdx.x+1 : elt_n%gridDim.x;
  intptr_t i = i0 + threadIdx.x;
  while(i < i1) {
    elts[i] = genInput<T>(op, rank_n, rank_me, seed, elt_ix0+i);
    #if 0
    T output = genOutput<T>(op, rank_n, seed, elt_ix0+i);
    printf("prepareInput2 T=%d seed=0x%llx r=%d ix=%lld x=%g output=%g elts=%p\n",
      std::is_same<T,int>::value, (long long)seed, int(rank_me), (long long)i, (float)elts[i], (float)output, elts);
    #endif
    i += blockDim.x;
  }
}

template<typename ReduceOp>
void prepareInput1(
    void *elts, intptr_t elt_n, int elt_ty, ReduceOp op, int rank_n, int rank_me,
    uint64_t seed, intptr_t elt_ix0, hipStream_t stream
  ) {
  int block_n = std::min<intptr_t>(32, (elt_n + 4*512-1)/(4*512));
  #define CASE_TY(T) prepareInput2<<<block_n, 512, 0, stream>>>((T*)elts, elt_n, op, rank_n, rank_me, seed, elt_ix0); break;
  switch(elt_ty) {
  case ncclInt8: CASE_TY(int8_t)
  case ncclUint8: CASE_TY(uint8_t)
  case ncclInt32: CASE_TY(int32_t)
  case ncclUint32: CASE_TY(uint32_t)
  case ncclInt64: CASE_TY(int64_t)
  case ncclUint64: CASE_TY(uint64_t)
  case ncclFloat16: CASE_TY(__half)
  #if HAVE_ncclBfloat16
  case ncclBfloat16: CASE_TY(__hip_bfloat16)
  #endif
  case ncclFloat32: CASE_TY(float)
  case ncclFloat64: CASE_TY(double)
  default: assert(0);
  }
  #undef CASE_TY
}
}

void ncclVerifiablePrepareInput(
    void *elts, intptr_t elt_n, int elt_ty, int red_op, int rank_n, int rank_me,
    uint64_t seed, intptr_t elt_ix0, hipStream_t stream
  ) {
  #define CASE_OP(op) \
    if(rank_n == 1) \
      prepareInput1(elts, elt_n, elt_ty, ReduceNil(), rank_n, rank_me, seed, elt_ix0, stream); \
    else \
      prepareInput1(elts, elt_n, elt_ty, op, rank_n, rank_me, seed, elt_ix0, stream); \
    break;
  switch(red_op) {
  case ncclSum: CASE_OP(ReduceSum())
  case ncclMin: CASE_OP(ReduceMin())
  case ncclMax: CASE_OP(ReduceMax())
  case ncclProd: CASE_OP(ReduceProd())
  #if HAVE_ncclAvg
  case ncclAvg: CASE_OP(ReduceAvg{rank_n})
  #endif
  #if HAVE_ncclPreMulSum
  default: CASE_OP(ReducePreMulSum())
  #endif
  }
  #undef CASE_OP
}
#endif

////////////////////////////////////////////////////////////////////////////////

#if !SELF_TEST
namespace {
template<typename T, typename ReduceFn>
__global__ void prepareExpected2(
    T *elts, intptr_t elt_n, ReduceFn op, int rank_n,
    uint64_t seed, intptr_t elt_ix0
  ) {
  intptr_t i0 = blockIdx.x*(elt_n/gridDim.x);
  i0 += blockIdx.x < elt_n%gridDim.x ? blockIdx.x : elt_n%gridDim.x;
  intptr_t i1 = (blockIdx.x+1)*(elt_n/gridDim.x);
  i1 += blockIdx.x+1 < elt_n%gridDim.x ? blockIdx.x+1 : elt_n%gridDim.x;
  intptr_t i = i0 + threadIdx.x;
  while(i < i1) {
    elts[i] = genOutput<T>(op, rank_n, seed, elt_ix0+i);
    #if 0
    printf("prepareExpected2 seed=0x%llx ix=%lld x=%g elts=%p\n",
      (long long)seed, (long long)(elt_ix0+i), (float)elts[i], elts);
    #endif
    i += blockDim.x;
  }
}

template<typename ReduceOp>
void prepareExpected1(
    void *elts, intptr_t elt_n, int elt_ty, ReduceOp op, int rank_n,
    uint64_t seed, intptr_t elt_ix0, hipStream_t stream
  ) {
  int block_n = std::min<intptr_t>(32, (elt_n + 4*512-1)/(4*512));
  #define CASE_TY(T) prepareExpected2<<<block_n, 512, 0, stream>>>((T*)elts, elt_n, op, rank_n, seed, elt_ix0); break;
  switch(elt_ty) {
  case ncclInt8: CASE_TY(int8_t)
  case ncclUint8: CASE_TY(uint8_t)
  case ncclInt32: CASE_TY(int32_t)
  case ncclUint32: CASE_TY(uint32_t)
  case ncclInt64: CASE_TY(int64_t)
  case ncclUint64: CASE_TY(uint64_t)
  case ncclFloat16: CASE_TY(__half)
  #if HAVE_ncclBfloat16
  case ncclBfloat16: CASE_TY(__hip_bfloat16)
  #endif
  case ncclFloat32: CASE_TY(float)
  case ncclFloat64: CASE_TY(double)
  default: assert(0);
  }
  #undef CASE_TY
}
}

void ncclVerifiablePrepareExpected(
    void *elts, intptr_t elt_n, int elt_ty, int red_op, int rank_n,
    uint64_t seed, intptr_t elt_ix0, hipStream_t stream
  ) {
  #define CASE_OP(op) \
    if(rank_n == 1) \
      prepareExpected1(elts, elt_n, elt_ty, ReduceNil(), rank_n, seed, elt_ix0, stream); \
    else \
      prepareExpected1(elts, elt_n, elt_ty, op, rank_n, seed, elt_ix0, stream); \
    break;
  switch(red_op) {
  case ncclSum: CASE_OP(ReduceSum())
  case ncclMin: CASE_OP(ReduceMin())
  case ncclMax: CASE_OP(ReduceMax())
  case ncclProd: CASE_OP(ReduceProd())
  #if HAVE_ncclAvg
  case ncclAvg: CASE_OP(ReduceAvg{rank_n})
  #endif
  #if HAVE_ncclPreMulSum
  default: CASE_OP(ReducePreMulSum())
  #endif
  }
  #undef CASE_OP
}
#endif

////////////////////////////////////////////////////////////////////////////////

namespace {
/* How we compare floating point values when exactness is impossible is interesting.
 * First, we take note that simply reinterpreting integer bits as floating point
 * gives us a monotonic mapping which exponentially spaces out floats. Thus
 * consecutive integers encode consecutive floats. In general, using integer
 * subraction on the bitpatterns of two floats gives us an integer which is the
 * logarithm of their relative difference. But, if the floats always have similar
 * exponents, than the integer difference is actually proportional to the
 * relative error (this is because we are counting hops in the mantissa bits only,
 * not the exponent bits). So a cheap way to compare if two floats are relatively
 * close is: abs(intBits(a), intBits(b)) < tolerance. The following formula
 * calculates such a tolerance for a summation of n floats. This formula
 * was derived by inspecting the maximum observed integer difference over many
 * random runs of summation. The parameter values were computed by the
 * companion program "inexact_regress.cu".
 */
__host__ __device__ unsigned calcSumFloatTolerance(int rank_n, int elt_ty) {
  float power, coef;
  switch(elt_ty) {
  case ncclFloat32:
  case ncclFloat64:
    power = .51f;
    coef = 1.25f;
    break;
  case ncclFloat16:
    power = .91f;
    coef = .75f;
    break;
  #if HAVE_ncclBfloat16
  case ncclBfloat16:
    power = .91f;
    coef = .66f;
    break;
  #endif
  }
  #if __CUDA_ARCH__
    return 1 + unsigned(coef*powf(float(rank_n), power));
  #else
    return 1 + unsigned(coef*std::pow(float(rank_n), power));
  #endif
}

template<typename T>
__host__ __device__  uint64_t calcDelta(T a, T b) {
  union { T t; uint8_t i1; uint16_t i2; uint32_t i4; uint64_t i8; } x, y;
  x.t = a;
  y.t = b;
  switch(sizeof(T)) {
  case 1:  return x.i1 < y.i1 ? y.i1 - x.i1 : x.i1 - y.i1;
  case 2:  return x.i2 < y.i2 ? y.i2 - x.i2 : x.i2 - y.i2;
  case 4:  return x.i4 < y.i4 ? y.i4 - x.i4 : x.i4 - y.i4;
  default: return x.i8 < y.i8 ? y.i8 - x.i8 : x.i8 - y.i8;
  }
}
}

////////////////////////////////////////////////////////////////////////////////

#if !SELF_TEST
namespace {
template<typename T>
__global__ void verifyPrepared(
    T const *results, T const *expected, intptr_t elt_n, unsigned tolerance, int64_t *bad_elt_n
  ) {
  intptr_t i0 = blockIdx.x*(elt_n/gridDim.x);
  i0 += blockIdx.x < elt_n%gridDim.x ? blockIdx.x : elt_n%gridDim.x;
  intptr_t i1 = (blockIdx.x+1)*(elt_n/gridDim.x);
  i1 += blockIdx.x+1 < elt_n%gridDim.x ? blockIdx.x+1 : elt_n%gridDim.x;
  intptr_t i = i0 + threadIdx.x;
  int64_t bad = 0;

  while(i < i1) {
    T a = results[i], b = expected[i];
    T delta = a < b ? b - a : a - b;
    bad += tolerance < delta ? 1 : 0;
    #if 0
      if(tolerance < delta) {
        printf("verifyPrepared ix=%lld got=%g exp=%g\n", (long long)i, (float)results[i], (float)expected[i]);
      }
    #endif
    i += blockDim.x;
  }
  //asm volatile("red.global.add.u64 [%0],%1;" :: "l"(bad_elt_n), "l"(bad));
  atomicAdd((unsigned long *)bad_elt_n, (unsigned long)bad);
}

template<typename T, typename Uint, typename ReduceFn>
__global__ void verifyInline2(
    T const *results, intptr_t elt_n, ReduceFn op, int rank_n, uint64_t seed,
    intptr_t elt_ix0, unsigned tolerance, int64_t *bad_elt_n
  ) {
  intptr_t i0 = blockIdx.x*(elt_n/gridDim.x);
  i0 += blockIdx.x < elt_n%gridDim.x ? blockIdx.x : elt_n%gridDim.x;
  intptr_t i1 = (blockIdx.x+1)*(elt_n/gridDim.x);
  i1 += blockIdx.x+1 < elt_n%gridDim.x ? blockIdx.x+1 : elt_n%gridDim.x;
  intptr_t i = i0 + threadIdx.x;
  int64_t bad = 0;

  while(i < i1) {
    union { T t; Uint u; } a, b;
    a.t = results[i];
    b.t = genOutput<T>(op, rank_n, seed, elt_ix0+i);
    Uint delta = a.u < b.u ? b.u - a.u : a.u - b.u;
    bad += tolerance < delta ? 1 : 0;
    #if 0
      T input = genInput<T>(op, rank_n, 0, seed, elt_ix0+i);
      if(tolerance < delta) {
        printf("verifyInline2 fail T=%d ix=%lld got=%g exp=%g input=%g\n",
          std::is_same<T,int>::value, (long long)i, (float)a.t, (float)b.t, (float)input);
      } else {
        printf("verifyInline2 pass T=%d ix=%lld got=%g exp=%g input=%g\n",
          std::is_same<T,int>::value, (long long)i, (float)a.t, (float)b.t, (float)input);
      }
    #endif
    i += blockDim.x;
  }
  //asm volatile("red.global.add.u64 [%0],%1;" :: "l"(bad_elt_n), "l"(bad));
  atomicAdd((unsigned long*)bad_elt_n, (unsigned long)bad);
}

template<typename T, typename Uint>
void verifyInline1(
    T const *results, intptr_t elt_n, int red_op, int rank_n, uint64_t seed, intptr_t elt_ix0,
    unsigned tolerance, int64_t *bad_elt_n, hipStream_t stream, int block_n
  ) {
  #define CASE_OP(op) \
    if(rank_n == 1) \
    verifyInline2<T, Uint><<<block_n, 512, 0, stream>>> \
      ((T const*)results, elt_n, ReduceNil(), rank_n, seed, elt_ix0, tolerance, bad_elt_n); \
    else \
    verifyInline2<T, Uint><<<block_n, 512, 0, stream>>> \
      ((T const*)results, elt_n, op, rank_n, seed, elt_ix0, tolerance, bad_elt_n); \
    break;
  switch(red_op) {
  case ncclSum: CASE_OP(ReduceSum())
  case ncclMin: CASE_OP(ReduceMin())
  case ncclMax: CASE_OP(ReduceMax())
  case ncclProd: CASE_OP(ReduceProd())
  #if HAVE_ncclAvg
  case ncclAvg: CASE_OP(ReduceAvg{rank_n})
  #endif
  #if HAVE_ncclPreMulSum
  default: CASE_OP(ReducePreMulSum())
  #endif
  }
  #undef CASE_OP
}
}

void ncclVerifiableVerify(
    void const *results, void const *expected, intptr_t elt_n, int elt_ty,
    int red_op, int rank_n, uint64_t seed, intptr_t elt_ix0,
    int64_t *bad_elt_n, hipStream_t stream
  ) {
  bool floating = elt_ty == ncclFloat16 || elt_ty == ncclFloat32 || elt_ty == ncclFloat64;
  #if HAVE_ncclBfloat16
    floating |= elt_ty == ncclBfloat16;
  #endif

  unsigned tolerance = 0;
  #if HAVE_ncclAvg
  if (floating && red_op == ncclAvg)
    tolerance = calcSumFloatTolerance(rank_n, elt_ty);
  #endif

  int block_n = std::min<intptr_t>(32, (elt_n + 4*512-1)/(4*512));

  *bad_elt_n = 0;
  #define CASE_TY(T, Uint) { \
      if(expected != nullptr) { \
        verifyPrepared<<<block_n, 512, 0, stream>>>((Uint const*)results, (Uint const*)expected, elt_n, tolerance, bad_elt_n); \
      } else { \
        verifyInline1<T, Uint>((T const*)results, elt_n, red_op, rank_n, seed, elt_ix0, tolerance, bad_elt_n, stream, block_n); \
      } \
    } break;
  switch(elt_ty) {
  case ncclInt8: CASE_TY(int8_t, uint8_t)
  case ncclUint8: CASE_TY(uint8_t, uint8_t)
  case ncclInt32: CASE_TY(int32_t, uint32_t)
  case ncclUint32: CASE_TY(uint32_t, uint32_t)
  case ncclInt64: CASE_TY(int64_t, uint64_t)
  case ncclUint64: CASE_TY(uint64_t, uint64_t)
  case ncclFloat16: CASE_TY(__half, uint16_t)
  #if HAVE_ncclBfloat16
  case ncclBfloat16: CASE_TY(__hip_bfloat16, uint16_t)
  #endif
  case ncclFloat32: CASE_TY(float, uint32_t)
  case ncclFloat64: CASE_TY(double, uint64_t)
  default: assert(0);
  }
  #undef CASE_TY
}
#endif

////////////////////////////////////////////////////////////////////////////////

#if SELF_TEST
#include <iostream>

template<typename T, typename Op>
__device__ void sweep2(int ty, char const *tyname, Op op, char const *opname, int rank_n) {
  //if(!std::is_same<T,half>::value) return;
  //if(!std::is_same<Op,ReduceProd>::value) return;
  //if(rank_n!=3) return;

  unsigned tolerance = !IsIntegral<T>::value && std::is_same<Op,ReduceAvg>::value ? calcSumFloatTolerance(rank_n, ty) : 0;
  uint64_t seed = 0xc8e2bed69766d533;

  for(int ix=threadIdx.x; ix < 10000; ix+=blockDim.x) {
    //if(ix!=387) continue;
    T y = genOutput<T>(op, rank_n, seed, ix);
    T sum;
    for(int r=0; r < rank_n; r++) {
      T x = genInput<T>(op, rank_n, r, seed, ix);
      x = op.preOp(x, r);
      sum = r==0 ? x : op(sum, inhibit(x));
      //std::printf("x = %llx, sum = %llx\n", bitsOf(x), bitsOf(sum));
    }
    sum = op.postOp(sum);
    if(tolerance < calcDelta(sum, y)) {
      printf(
        //"%10g != %10g  :  T=%-8s op=%-9s rank_n=%-1d ix=%-1d\n",
        "%llx != %llx  :  T=%-8s op=%-9s rank_n=%-1d ix=%-1d\n",
        *(long long*)&sum, *(long long*)&y, tyname, opname, rank_n, ix
      );
    }
  }
}

template<typename T>
__device__ void sweep1(int ty, char const *tyname) {
  for(int i=0; i < 10; i++) {
    int rank_n = (1<<i) + i;
    sweep2<T>(ty, tyname, ReduceSum(), "sum", rank_n);
    sweep2<T>(ty, tyname, ReduceProd(), "prod", rank_n);
    sweep2<T>(ty, tyname, ReduceMin(), "min", rank_n);
    sweep2<T>(ty, tyname, ReduceMax(), "max", rank_n);
    sweep2<T>(ty, tyname, ReducePreMulSum(), "premulsum", rank_n);
    sweep2<T>(ty, tyname, ReduceAvg{rank_n}, "avg", rank_n);
  }
}

__global__ void sweep() {
  sweep1<int8_t>(ncclInt8, "int8");
  sweep1<uint8_t>(ncclUint8, "uint8");
  sweep1<int32_t>(ncclInt32, "int32");
  sweep1<uint32_t>(ncclUint32, "uint32");
  sweep1<int64_t>(ncclInt64, "int64");
  sweep1<uint64_t>(ncclUint64, "uint64");
  sweep1<__half>(ncclFloat16, "half");
  #if HAVE_ncclBfloat16
    sweep1<__hip_bfloat16>(ncclBfloat16, "bfloat16");
  #endif
  sweep1<float>(ncclFloat32, "float");
  sweep1<double>(ncclFloat64, "double");
}

int main(int arg_n, char **args) {
  std::cerr<<"You are hoping to see no output beyond this line."<<std::endl;
  hipSetDevice(0);
  sweep<<<1,512>>>();
  hipDeviceSynchronize();
  return 0;
}
#endif
